#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


/* Shorthand for less typing */
typedef unsigned char uchar;

/* Declarations of output functions */
void output(int* pixel);
void fancycolour(uchar *p, int iter);
void savebmp(char *name, uchar *buffer, int x, int y);

/* Struct for complex numbers */
typedef struct {
    float real, imag;
} complex_t;

/* Size of image, in pixels */
const int XSIZE = 2560;
const int YSIZE = 2048;

/* Divide the problem into blocks of BLOCKX X BLOCKY threads */
const int BLOCKY = 8;
const int BLOCKX = 8;

/* Max number of iterations */
const int MAXITER = 255;

/* Range in x direction */
const float xleft = -2.0;
const float xright = 1.0;
const float ycenter = 0.0;

/* Range in y direction, calculated in main
 * based on range in x direction and image size
 */
float yupper, ylower;

/* Distance between numbers */
float step;


/* Timing */
double walltime() {
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}


/* Error handling function */
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



/* Actual GPU kernel which will be executed in parallel on the GPU */
__global__ void mandel_kernel( int* pixel, float step , float ylower ){

    // Find matrix indices
    int i = blockIdx.x * BLOCKX + threadIdx.x;
    int j = blockIdx.y * BLOCKY + threadIdx.y;

    // Initialize iteration variable
    int iter = 0;

    // Initalize complex floats
    complex_t c, z, temp;

    // Set inital value for c
    c.real = ( xleft + step * i );
    c.imag = ( ylower + step * j );

    // Copy to z
    z = c;

    // Initalize control varibles
    float upperLimit = 4;
    float twoTimes = 2;

    // Loop through and iterate to display Mandelbrot
    while( z.real * z.real + z.imag * z.imag < upperLimit) {
        temp.real = z.real * z.real - z.imag * z.imag + c.real;
        temp.imag = twoTimes * z.real * z.imag + c.imag;

        z = temp;

        iter++;

        // Stop iteration when max is reached
        if (iter == MAXITER){
            break;
        }
    }
    // Place iteration result in pixel array
    pixel[j * XSIZE + i] = iter;

}

/* Set up and call GPU kernel */
void calculate_cuda(int* pixel){

    // Allocate device memory
    int *device_pixel;
    HANDLE_ERROR(hipMalloc((void**) &device_pixel,XSIZE * YSIZE * sizeof(int) ));

    // Copy from CPU to GPU memory
    HANDLE_ERROR(hipMemcpy(device_pixel,pixel,sizeof(int) * XSIZE * YSIZE , hipMemcpyHostToDevice));

    // Compute thread-block size
    dim3 gridBlock(XSIZE/BLOCKX , YSIZE/BLOCKY);
    dim3 threadBlock(BLOCKX, BLOCKY);

    // Call mandelbrot kernel
    mandel_kernel<<<gridBlock,threadBlock>>>(device_pixel , step ,ylower);

    // Transfer result from GPU to CPU
    HANDLE_ERROR(hipMemcpy(pixel, device_pixel , sizeof(int) * XSIZE * YSIZE , hipMemcpyDeviceToHost));

    // Free GPU memory
    HANDLE_ERROR(hipFree(device_pixel));

}


/* Calculate the number of iterations until divergence fdoubleor each pixel.
 * If divergence never happens, return MAXITER
 */
void calculate(int* pixel) {
    for (int i = 0; i < XSIZE; i++) {
        for (int j = 0; j < YSIZE; j++) {
            complex_t c, z, temp;
            int iter = 0;
            c.real = (xleft + step * i);
            c.imag = (ylower + step * j);
            z = c;
            while (z.real * z.real + z.imag * z.imag < 4) {
                temp.real = z.real * z.real - z.imag * z.imag + c.real;
                temp.imag = 2 * z.real * z.imag + c.imag;
                z = temp;
                iter++;
                if(iter == MAXITER){
                    break;
                }
            }
            pixel[j * XSIZE + i] = iter;
        }
    }
}


int main(int argc, char **argv) {

    /* Check input arguments */
    if (argc == 1) {
        puts("Usage: MANDEL n");
        puts("n decides whether image should be written to disk (1 = yes, 0 = no)");
        return 0;
    }

    /* Find number of CUDA devices (GPUs)
     * and print the name of the first one.
     */
    int n_devices;
    hipGetDeviceCount(&n_devices);
    printf("Number of CUDA devices: %d\n", n_devices);
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    printf("CUDA device name: %s\n" , device_prop.name);

    /* Calculate the range in the y - axis such that we preserve the aspect ratio */
    step = (xright - xleft)/XSIZE;
    yupper = ycenter + (step * YSIZE)/2;
    ylower = ycenter - (step * YSIZE)/2;

    /* Global arrays for iteration counts/pixels
     * One array for the result of the CPU calculation,
     * one for the result of the GPU calculation.
     * (Both are in the host/CPU memory)
     */
    int* pixel_for_cpu = (int*) malloc(sizeof(int) * XSIZE * YSIZE);
    int* pixel_for_gpu = (int*) malloc(sizeof(int) * XSIZE * YSIZE);


    /* Perform calculation on CPU */
    double start_cpu = walltime();
    calculate(pixel_for_cpu);
    double end_cpu = walltime();

    /* Perform calculations on GPU */
    double start_gpu = walltime();
    calculate_cuda(pixel_for_gpu);
    double end_gpu = walltime();

    /* Compare execution times
     * The GPU time also includes the time for memory allocation and transfer
     */
    printf("CPU time: %f s\n" , (end_cpu-start_cpu));
    printf("GPU time: %f s\n" , (end_gpu-start_gpu));


    /* Output */
    if (strtol(argv[1], NULL, 10) != 0) {
        output(pixel_for_gpu);
    }

    return 0;
}

/* Save 24 - bits bmp file, buffer must be in bmp format: upside - down */
void savebmp(char *name, uchar *buffer, int x, int y) {
    FILE *f = fopen(name, "wb");
    if (!f) {
        printf("Error writing image to disk.\n");
        return;
    }
    unsigned int size = x * y * 3 + 54;
    uchar header[54] = {'B', 'M',
        size&255,
        (size >> 8)&255,
        (size >> 16)&255,
        size >> 24,
        0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0, x&255, x >> 8, 0,
        0, y&255, y >> 8, 0, 0, 1, 0, 24, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    fwrite(header, 1, 54, f);
    fwrite(buffer, 1, XSIZE * YSIZE * 3, f);
    fclose(f);
}

/* Given iteration number, set a colour */
void fancycolour(uchar *p, int iter) {
    if (iter == MAXITER);
    else if (iter < 8) { p[0] = 128 + iter * 16; p[1] = p[2] = 0; }
    else if (iter < 24) { p[0] = 255; p[1] = p[2] = (iter - 8) * 16; }
    else if (iter < 160) { p[0] = p[1] = 255 - (iter - 24) * 2; p[2] = 255; }
    else { p[0] = p[1] = (iter - 160) * 2; p[2] = 255 - (iter - 160) * 2; }
}

/* Create nice image from iteration counts. take care to create it upside down (bmp format) */
void output(int* pixel){
    unsigned char *buffer = (unsigned char*)calloc(XSIZE * YSIZE * 3, 1);
    for (int i = 0; i < XSIZE; i++) {
        for (int j = 0; j < YSIZE; j++) {
            int p = ((YSIZE - j - 1) * XSIZE + i) * 3;
            fancycolour(buffer + p, pixel[(i + XSIZE * j)]);
        }
    }
    /* write image to disk */
    savebmp("mandel2.bmp", buffer, XSIZE, YSIZE);
    free(buffer);
}
